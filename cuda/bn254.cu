// Copyright Supranational LLC
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0

#include <hip/hip_runtime.h>

#include <ec/jacobian_t.hpp>
#include <ec/xyzz_t.hpp>

#include <ff/alt_bn128.hpp>

typedef jacobian_t<fp_t> point_t;
typedef xyzz_t<fp_t> bucket_t;
typedef bucket_t::affine_t affine_t;
typedef fr_t scalar_t;

#include <msm/pippenger.cuh>
#include <spmvm/spmvm.cuh>

#ifndef __CUDA_ARCH__
extern "C" 
RustError cuda_sparse_matrix_mul_bn254(spmvm_host_t<scalar_t> *csr, const witness_t<scalar_t> *witness, 
                                        scalar_t *out, size_t nthreads)
{
    return sparse_matrix_mul<scalar_t>(csr, witness, out, nthreads);
}

extern "C"
RustError cuda_pippenger_bn254(point_t *out, const affine_t points[], size_t npoints,
                                              const scalar_t scalars[])
{   return mult_pippenger<bucket_t>(out, points, npoints, scalars);   }
#endif
