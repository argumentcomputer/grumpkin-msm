// Copyright Supranational LLC
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0

#include <hip/hip_runtime.h>

#include <ec/jacobian_t.hpp>
#include <ec/xyzz_t.hpp>

#include <ff/alt_bn128.hpp>

typedef jacobian_t<fp_t> point_t;
typedef xyzz_t<fp_t> bucket_t;
typedef bucket_t::affine_t affine_t;
typedef fr_t scalar_t;

#include <msm/pippenger.cuh>

#ifndef __CUDA_ARCH__

extern "C" void drop_msm_context_bn254(msm_context_t<affine_t::mem_t> &ref) {
    CUDA_OK(hipFree(ref.d_points));
}

extern "C" RustError
cuda_bn254_init(const affine_t points[], size_t npoints, msm_context_t<affine_t::mem_t> *msm_context)
{
    return mult_pippenger_init<bucket_t, point_t, affine_t, scalar_t>(points, npoints, msm_context);
}

extern "C" RustError cuda_bn254(point_t *out, const affine_t points[], size_t npoints,
                                          const scalar_t scalars[])
{
    return mult_pippenger<bucket_t>(out, points, npoints, scalars);
}

extern "C" RustError cuda_bn254_with(point_t *out, msm_context_t<affine_t::mem_t> *msm_context,
                                                       const scalar_t scalars[])
{
    return mult_pippenger_with<bucket_t, point_t, affine_t, scalar_t>(out, msm_context, scalars);
}
#endif
