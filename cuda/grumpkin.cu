// Copyright Supranational LLC
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0

#include <hip/hip_runtime.h>

#include <ec/jacobian_t.hpp>
#include <ec/xyzz_t.hpp>

#include <ff/alt_bn128.hpp>

typedef jacobian_t<fr_t> point_t;
typedef xyzz_t<fr_t> bucket_t;
typedef bucket_t::affine_t affine_t;
typedef fp_t scalar_t;

#include <msm/pippenger.cuh>

#ifndef __CUDA_ARCH__
extern "C"
RustError cuda_pippenger_grumpkin(point_t *out, const affine_t points[], size_t npoints,
                                              const scalar_t scalars[])
{   return mult_pippenger<bucket_t>(out, points, npoints, scalars);   }
#endif
