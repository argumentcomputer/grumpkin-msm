// Copyright Supranational LLC
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0

#include <hip/hip_runtime.h>

#include <ec/jacobian_t.hpp>
#include <ec/xyzz_t.hpp>

#include <ff/pasta.hpp>

typedef jacobian_t<vesta_t> point_t;
typedef xyzz_t<vesta_t> bucket_t;
typedef bucket_t::affine_t affine_t;
typedef pallas_t scalar_t;

#include <msm/pippenger.cuh>

#ifndef __CUDA_ARCH__

extern "C" void drop_msm_context_vesta(msm_context_t<affine_t::mem_t> &ref)
{
    CUDA_OK(hipFree(ref.d_points));
}

extern "C" RustError
cuda_vesta_init(const affine_t points[], size_t npoints, msm_context_t<affine_t::mem_t> *msm_context)
{
    return mult_pippenger_init<bucket_t, point_t, affine_t, scalar_t>(points, npoints, msm_context);
}

extern "C" RustError cuda_vesta(point_t *out, const affine_t points[], size_t npoints,
                                const scalar_t scalars[], size_t nscalars)
{
    return mult_pippenger<bucket_t>(out, points, npoints, scalars, nscalars);
}

extern "C" RustError cuda_vesta_with(point_t *out, msm_context_t<affine_t::mem_t> *msm_context, size_t npoints,
                                     const scalar_t scalars[], size_t nscalars, uint32_t pidx[])
{
    return mult_pippenger_with<bucket_t, point_t, affine_t, scalar_t>(out, msm_context, npoints, scalars, nscalars, pidx);
}

#endif
